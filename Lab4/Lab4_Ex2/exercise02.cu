#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);



__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

void vectorAddCPU(int* a, int* b, int* c, int max) {
	for (int i = 0; i < max; i++) {
		c[i] = a[i] + b[i];
	}
}

int validate(int* test, int* ref, int max) {
	int err_count = 0;
	for (int i = 0; i < max; i++) {
		int x = test[i];
		int y = ref[i];
		if (x != y) {
			err_count++;
			printf("ERROR at index %d! Expected: %d; Got: %d\n", i, y, x);
		}
	}
	return err_count;
}


int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
	int blocks = N / THREADS_PER_BLOCK;
	if (blocks * THREADS_PER_BLOCK < N) blocks++;
	vectorAdd <<< blocks, THREADS_PER_BLOCK >>> (d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	// Check!
	vectorAddCPU(a, b, c_ref, N);
	errors = validate(c, c_ref, N);
	if (errors > 0) {
		printf("Error count: %d\n", errors);
	}

	// Cleanup
	free(a); free(b); free(c);
	free(c_ref);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
